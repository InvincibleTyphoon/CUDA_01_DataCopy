
#include <hip/hip_runtime.h>
#include <cstdio>

int main(void)
{
	const int  SIZE = 5;
	const int a[SIZE] = { 1,2,3,4,5 };
	int b[SIZE] = { 0,0,0,0,0 };

	//시작 전 값 확인
	printf("a = {%d,%d,%d,%d,%d}\n", a[0], a[1], a[2], a[3], a[4]);

	int* dev_a = 0;
	int* dev_b = 0;

	//GPU 메모리 공간 할당
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));

	//RAM -> VRAM 메모리 복사
	//a -> dev_a
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	//dev_a -> dev_b
	hipMemcpy(dev_b, dev_a, SIZE * sizeof(int), hipMemcpyDeviceToDevice);
	//dev_b -> b
	hipMemcpy(b, dev_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//VRAM 할당 공간 해제
	hipFree(dev_a);
	hipFree(dev_b);

	printf("b = {%d,%d,%d,%d,%d}\n", b[0], b[1], b[2], b[3], b[4]);

	return 0;
}