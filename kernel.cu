
#include <hip/hip_runtime.h>
#include <cstdio>

int main(void)
{
	const int  SIZE = 5;
	const int a[SIZE] = { 1,2,3,4,5 };
	int b[SIZE] = { 0,0,0,0,0 };

	//���� �� �� Ȯ��
	printf("a = {%d,%d,%d,%d,%d}\n", a[0], a[1], a[2], a[3], a[4]);

	int* dev_a = 0;
	int* dev_b = 0;

	//GPU �޸� ���� �Ҵ�
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));

	//RAM -> VRAM �޸� ����
	//a -> dev_a
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	//dev_a -> dev_b
	hipMemcpy(dev_b, dev_a, SIZE * sizeof(int), hipMemcpyDeviceToDevice);
	//dev_b -> b
	hipMemcpy(b, dev_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//VRAM �Ҵ� ���� ����
	hipFree(dev_a);
	hipFree(dev_b);

	printf("b = {%d,%d,%d,%d,%d}\n", b[0], b[1], b[2], b[3], b[4]);

	return 0;
}